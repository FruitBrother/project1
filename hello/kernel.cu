#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>


using namespace std;

__global__ void scan_bel(int* inputarray, int loop, int* outputarray, int number)
{
	unsigned int thIdx = blockIdx.x * blockDim.x + threadIdx.x;

	int divisor = 2;
	int adder = 1;
	int temp;
	for (int i = 0; i<loop; i++)
	{
		if (thIdx % (divisor) == divisor - 1)
		{
			outputarray[thIdx] = outputarray[thIdx - adder] + outputarray[thIdx];
			if (thIdx == 255)
				printf("outputarray[%d] = %d\n", thIdx, outputarray[thIdx]);
		}
		__syncthreads();
		divisor *= 2;
		adder *= 2;
	}

	divisor = number;
	adder = divisor / 2;
	outputarray[number - 1] = 0;
	for (int i = 0; i<loop; i++)
	{
		if (thIdx % (divisor) == divisor - 1)
		{
			temp = outputarray[thIdx];
			outputarray[thIdx] = outputarray[thIdx - adder] + outputarray[thIdx];
			//printf("outputarray[%d] = %d | ", thIdx, outputarray[thIdx]);
			outputarray[thIdx - adder] = temp;
			//printf("outputarray[%d] = %d\n", thIdx - adder, temp);
		}
		__syncthreads();
		divisor /= 2;
		adder /= 2;
	}
}
int main()
{
	int number = 256;
	int* arrayin;
	int* arrayin_d;
	int* arrayout;
	int* arrayout_d;


	arrayin = new int[number];// {1, 1, 0, 1, 0, 0, 0, 1};
	arrayout = new int[number];

	for (int i = 0; i<number; i++)
	{
		arrayin[i] = i;
	}

	int loop;
	loop = (int)log2f(number);

	hipMalloc((void**)&arrayin_d, number * sizeof(int));
	hipMalloc((void**)&arrayout_d, number * sizeof(int));

	hipMemcpy(arrayin_d, arrayin, number * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(arrayout_d, arrayin, number * sizeof(int), hipMemcpyHostToDevice);

	scan_bel <<<1, number>>>(arrayin_d, loop, arrayout_d, number);

	hipMemcpy(arrayout, arrayout_d, number * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i<number; i++)
	{
		cout << arrayout[i] << endl;
	}
	delete[] arrayout;
	//delete[] arrayin;

	hipFree(arrayin_d);
	hipFree(arrayout_d);

	system("pause");

	return 0;
}