#include "hip/hip_runtime.h"
#pragma warning(disable:4819)
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include "hip/hip_runtime.h"
#include ""
#include "head.h"
#include "scan.h"
#define HASHSIZE 10000;
__global__ void createNeta(int *data, int step, int datanum, dint *equal, int equalsize, int* hash)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int i = 0;
	if (x >= datanum) return;
	unsigned int sum = 0;
	for (i = 0; i < equalsize; i++) {
		sum += data[x*step + equal[i].a];
	}
	sum %= HASHSIZE;
	if (hash[sum] = 0)
		atomicAdd(hash[sum], 1);//1
}
__global__ void createNetb(int *data, int step, int datanum, dint *equal, int equalsize, int* hash)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int i = 0;
	if (x >= datanum) return;
	unsigned int sum = 0;
	for (i = 0; i < equalsize; i++) {
		sum += data[x*step + equal[i].a];
	}
	sum %= HASHSIZE;
	if (hash[sum] != 0){
		atomicAdd(hash[sum], 1);//>1
	}
}
__global__ void useNeta(int *data, int step, int datanum, dint *equal, int equalsize, int* hash, int *res) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int i = 0;
	if (x >= datanum) return;
	unsigned int sum = 0;
	for (i = 0; i < equalsize; i++) {
		sum += data[x*step + equal[i].a];
	}
	sum %= HASHSIZE;
	if (hash[sum] >1 )
		res[x*step]
}

extern "C" void filter(int *rqa, int *rqb, dint* equal, int stepa, int stepb, int numa, int numb, int equalsize)
{
	int *dev_rqa = 0;
	int *dev_rqb = 0;
	dint *dev_equal = 0;
	int *d_hash = 0;
	int *res = 0;
	hipSetDevice(0);
	hipError_t cudaStatus;
	CUDACHECK(hipMalloc((void**)&dev_rqa, numa*stepa * sizeof(int)));
	CUDACHECK(hipMalloc((void**)&dev_rqb, numb*stepb * sizeof(int)));
	CUDACHECK(hipMalloc((void**)&dev_equal, sizeof(dint)*MAX(numa, numb)));
	CUDACHECK(hipMalloc((void**)&d_hash, sizeof(int)*numa));//a is min

	CUDACHECK(hipMemcpy(dev_rqa, rqa, numa*stepa * sizeof(int), hipMemcpyHostToDevice));
	CUDACHECK(hipMemcpy(dev_rqb, rqb, numb*stepb * sizeof(int), hipMemcpyHostToDevice));
	CUDACHECK(hipMemcpy(dev_equal, equal, sizeof(dint)*MAX(numa, numb), hipMemcpyHostToDevice));

	createNeta <<< (numa + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>> (dev_rqa, stepa, numa, dev_equal, equalsize, d_hash);
	createNetb <<< (numb + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>> (dev_rqb, stepb, numb, dev_equal, equalsize, d_hash);


}