#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#define MAX(a,b) (a<b)?b:a
struct dint {
	int a, b;
};

extern "C" void gpu(int *rqa, int *rqb, int* res, dint* equal, int stepa, int stepb, int resstep) {
	//allocate memory in gpu and tans the data to it
	int *dev_rqa = 0;
	int *dev_rqb = 0;
	int *dev_res = 0;
	int *dev_equal = 0;
	hipSetDevice(0);
	hipMalloc((void**)&dev_rqa, numa*stepa * sizeof(int));
	hipMalloc((void**)&dev_rqb, numb*stepb * sizeof(int));
	hipMalloc((void**)&dev_equal, sizeof(dint)*MAX(numa, numb));

	hipMemcpy(dev_rqa, rqa, numa*stepa * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_rqb, rqb, numb*stepb * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_res, res, numa*numb*resstep * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_equal, equal, sizeof(dint)*MAX(numa, numb), hipMemcpyHostToDevice);

	dim3 block(16, 16);
	dim3 grid((numa + 15) / 16, (numb + 15) / 16);
	join <<<grid, block >>> (dev_rqa, dev_rqb, dev_res, dev_equal, stepa, stepb, resstep, equalsize, numa, numb);
	hipDeviceSynchronize();
	hipMemcpy(res, dev_res, numa*numb*resstep * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_rqa);
	hipFree(dev_rqb);
	hipFree(dev_res);
	hipFree(dev_equal);
}


__global__ void join(int *rqa, int *rqb, int* res, dint* equal, int stepa, int stepb, int resstep, int equalsize, int numa, int numb)
{	int ea, eb;
	bool isMatch = true;
	int k, index, equalid, id;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockIdx.y + threadIdx.y;
	if (x >= numa || y >= numb) return;

	for (k = 0; k < equalsize; k++) {
		ea = equal[k].a + x*stepa;
		eb = equal[k].b + y*stepb;
		if (rqa[ea] != rqb[eb]) {
			isMatch = false;
			break;
		}
	}
	if (isMatch) {
		equalid = 0;
		index = 0;
		id = y*resstep + x;
		for (k = 0; k < stepa + stepb; k++) {
			if (k < stepa) {
				res[id + k] = rqa[x*stepa + k];
				index = k;
			}
			else if (equal[equalid].b != k - stepa) {
				res[id + (++index)] = rqb[y*stepb + (k - stepa)];
			}
			else {
				equalid++;
			}
		}
	}
}


// Helper function for using CUDA to add vectors in parallel.
void addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

	hipSetDevice(0);

	hipMalloc((void**)&dev_c, size * sizeof(int));
	hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    hipGetLastError();
    hipDeviceSynchronize();
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);


Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
}
