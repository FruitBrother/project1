#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include "hip/hip_runtime.h"
#include ""
#include "scan.h"
#include "head.h"
__global__ void scan_bel(unsigned int* inputarray, unsigned int loop, unsigned int* outputarray, int numa, int numb)
{
	unsigned int number = numa*numb;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= numa || y >= numb) return;
	unsigned int thIdx = y*numb + x;
	int divisor = 2;
	int adder = 1;
	unsigned int temp;

	for (int i = 0; i<loop; i++)
	{
		if (thIdx % (divisor) == divisor - 1)
		{
			outputarray[thIdx] = outputarray[thIdx - adder] + outputarray[thIdx];
			if (thIdx == 255)
				printf("outputarray[%d] = %d \n", thIdx, outputarray[thIdx]);
		}
		__syncthreads();
		divisor *= 2;
		adder *= 2;
	}

	divisor = number;
	adder = divisor / 2;

	outputarray[number - 1] = 0;
	for (int i = 0; i<loop; i++)
	{
		if (thIdx % (divisor) == divisor - 1)
		{
			temp = outputarray[thIdx];
			outputarray[thIdx] = outputarray[thIdx - adder] + outputarray[thIdx];
			outputarray[thIdx - adder] = temp;
		}
		__syncthreads();
		divisor /= 2;
		adder /= 2;
	}
	__syncthreads();
}
extern "C" void Scan(unsigned int * arrayin, unsigned int * arrayout, int numa, int numb)
{
	unsigned int number = numa*numb;
	unsigned int* arrayin_d = 0;
	unsigned int* arrayout_d = 0;
	int loop;
	loop = (int)log2f(number);
	hipSetDevice(0);

	hipMalloc((void**)&arrayin_d, number * sizeof(unsigned int));
	hipMalloc((void**)&arrayout_d, number * sizeof(unsigned int));

	hipMemcpy(arrayin_d, arrayin, number * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(arrayout_d, arrayin, number * sizeof(unsigned int), hipMemcpyHostToDevice);

	dim3 block(BLOCKX, BLOCKY);
	dim3 grid((numa + BLOCKX - 1) / BLOCKX, (numb + BLOCKY - 1) / BLOCKY);
	scan_bel <<<grid, block>>>(arrayin_d, loop, arrayout_d, numa, numb);
	hipDeviceSynchronize();

	hipMemcpy(arrayout, arrayout_d, number * sizeof(unsigned int), hipMemcpyDeviceToHost);

	hipFree(arrayin_d);
	hipFree(arrayout_d);
}
