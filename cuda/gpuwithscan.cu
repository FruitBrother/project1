#include "hip/hip_runtime.h"
#pragma warning(disable:4819)
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include "hip/hip_runtime.h"
#include ""
#include "head.h"
#include "scan.h"
__global__ void prejoin(int *rqa, int *rqb, unsigned int* flag, dint* equal, int stepa, int stepb, int equalsize, int numa, int numb)
{
	int ea, eb;
	bool isMatch = true;
	int k;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= numa || y >= numb) return;
	for (k = 0; k < equalsize; k++) {
		ea = equal[k].a + x*stepa;
		eb = equal[k].b + y*stepb;
		if (rqa[ea] != rqb[eb]) {
			isMatch = false;
			break;
		}
	}
	if (isMatch) {
		flag[x*numb + y] = 1;
	}
	else {
		flag[x*numb + y] = 0;
	}
}

__global__ void result_output(int *rqa, int *rqb, int* res, unsigned int* flag, unsigned int* flag_compute,dint* equal, int stepa, int stepb, int resstep, int numa, int numb)
{
	int k, index, equalid, id;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= numa || y >= numb) return;
	int ID = x*numb + y;
	if (flag[ID]) {
		equalid = 0;
		index = 0;
		//printf("(%d,%d)\n", x, y);
		id = ID*resstep;
		for (k = 0; k < stepa + stepb; k++) {
			if (k < stepa) {
				res[flag_compute[ID]*resstep + k] = rqa[x*stepa + k];

				index = k;
			}
			else if (equal[equalid].b != k - stepa) {
				res[flag_compute[ID]*resstep + (++index)] = rqb[y*stepb + (k - stepa)];
			}
			else {
				equalid++;
			}
		}
	}
}

extern "C" void gpuwithscan(int *rqa, int *rqb, int* res, dint* equal, int stepa, int stepb, int resstep, int numa, int numb, int equalsize, int* numresgpu)
{
	//allocate memory in gpu and tans the data to it
	int *dev_rqa = 0;
	int *dev_rqb = 0;
	int *dev_res = 0;
	unsigned int *dev_flag = 0;
	unsigned int *dev_flag_compute = 0;
	unsigned int *flag_compute = 0;
	dint *dev_equal = 0;
	hipError_t cudaStatus;
	hipSetDevice(0);

	CUDACHECK(hipMalloc((void**)&dev_rqa, numa*stepa * sizeof(int)));
	CUDACHECK(hipMalloc((void**)&dev_rqb, numb*stepb * sizeof(int)));
	CUDACHECK(hipMalloc((void**)&dev_flag, numa*numb * sizeof(unsigned int)));
	CUDACHECK(hipMalloc((void**)&dev_equal, sizeof(dint)*MAX(numa, numb)));

	CUDACHECK(hipMemcpy(dev_rqa, rqa, numa*stepa * sizeof(int), hipMemcpyHostToDevice));
	CUDACHECK(hipMemcpy(dev_rqb, rqb, numb*stepb * sizeof(int), hipMemcpyHostToDevice));
	CUDACHECK(hipMemcpy(dev_equal, equal, sizeof(dint)*MAX(numa, numb), hipMemcpyHostToDevice));

	dim3 block(BLOCKX, BLOCKY);
	dim3 grid((numa + BLOCKX - 1) / BLOCKX, (numb + BLOCKY - 1) / BLOCKY);
	
	//prejoin
	prejoin<<<grid, block>>>(dev_rqa, dev_rqb, dev_flag, dev_equal, stepa, stepb, equalsize, numa, numb);
	CUDAKERNELCHECK;
	//CUDACHECK(hipDeviceSynchronize());
	
	//get_result_info
	CUDACHECK(hipMalloc((void**)&dev_flag_compute, numa*numb * sizeof(unsigned int)));
	get_result_info(dev_flag_compute, dev_flag, numa*numb);
	//CUDACHECK(hipDeviceSynchronize());

	//allcate the result-- **may be more optimization needed here**
	flag_compute = (unsigned int*)malloc(sizeof(unsigned int)* numa*numb);
	CUDACHECK(hipMemcpy(flag_compute, dev_flag_compute, numa*numb * sizeof(unsigned int), hipMemcpyDeviceToHost));
	*numresgpu = flag_compute[numa*numb-1];
	free(flag_compute);
	CUDACHECK(hipMalloc((void**)&dev_res, (*numresgpu)*resstep * sizeof(int)));
	
	//output the result	
	result_output<<<grid, block>>>(dev_rqa, dev_rqb, dev_res, dev_flag, dev_flag_compute, dev_equal, stepa, stepb, resstep, numa, numb);
	CUDAKERNELCHECK;
	//CUDACHECK(hipDeviceSynchronize());

	CUDACHECK(hipMemcpy(res, dev_res, (*numresgpu)*resstep * sizeof(int), hipMemcpyDeviceToHost));

	hipFree(dev_flag);
	hipFree(dev_flag_compute);
	hipFree(dev_rqa);
	hipFree(dev_rqb);
	hipFree(dev_res);
	hipFree(dev_equal);	
}