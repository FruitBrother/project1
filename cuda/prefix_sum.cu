#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include "hip/hip_runtime.h"
#include ""
#include "head.h"
__global__ void work_efficient_scan_kernel(unsigned int *X, unsigned int *Y, int InputSize) {
	// XY[2*BLOCK_SIZE] is in shared memory
	__shared__ unsigned int XY[BLOCK_SIZE * 2];
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	//printf("%d=%d*%d+%d\n",i,blockIdx.x,blockDim.x,threadIdx.x);
	if (i < InputSize) {
		XY[threadIdx.x] = X[i];
		XY[threadIdx.x + BLOCK_SIZE] = 0;
	}
	printf("XY[%d]=%d\n", threadIdx.x, XY[threadIdx.x]);
	// the code below performs iterative scan on XY����
	for (unsigned int stride = 1; stride <= BLOCK_SIZE; stride *= 2) {
		__syncthreads();
		int index = (threadIdx.x + 1)*stride * 2 - 1;
		if (index < 2 * BLOCK_SIZE) {
			XY[index] += XY[index - stride];//index is alway bigger than stride
			printf("XY[%d] += XY[%d]= %d |\n", index , index - stride ,XY[index]);
		}
		__syncthreads();
	}
	// threadIdx.x+1 = 1,2,3,4....
	// stridek index = 1,3,5,7...


	for (unsigned int stride = BLOCK_SIZE / 2; stride > 0; stride /= 2) {
		__syncthreads();
		int index = (threadIdx.x + 1)*stride * 2 - 1;
		if (index < 2 * BLOCK_SIZE) {
			XY[index + stride] += XY[index];
			printf("XY[%d] += XY[%d]= %d |\n", index + stride, index, XY[index + stride]);
		}
	}
	__syncthreads();
	if (i < InputSize) Y[i] = XY[threadIdx.x];
}
extern "C" void prefix_sum(unsigned int *x, unsigned int *result, unsigned int size) {
	unsigned int* dev_x = 0;
	unsigned int* dev_result = 0;
	hipSetDevice(0);

	hipMalloc((void**)&dev_x, size * sizeof(unsigned int));
	hipMalloc((void**)&dev_result, size * sizeof(unsigned int));

	hipMemcpy(dev_x, x, size * sizeof(unsigned int), hipMemcpyHostToDevice);

	work_efficient_scan_kernel <<< 1, BLOCK_SIZE >>> (dev_x, dev_result, size);
	hipDeviceSynchronize();
	
	hipMemcpy(result, dev_result, size * sizeof(unsigned int), hipMemcpyDeviceToHost);

	hipFree(dev_result);
	hipFree(dev_x);
}