#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include "hip/hip_runtime.h"
#include ""
#include "head.h"
__global__ void work_efficient_scan_kernel(float *X, float *Y, int InputSize) {
	// XY[2*BLOCK_SIZE] is in shared memory
	_shared__ float XY[BLOCK_SIZE * 2];
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < InputSize) { XY[threadIdx.x] = X[i]; }

	// the code below performs iterative scan on XY����
	for (unsigned int stride = 1; stride <= BLOCK_SIZE; stride *= 2) {
		__syncthreads();
		int index = (threadIdx.x + 1)*stride * 2 - 1;
		if (index < 2 * BLOCK_SIZE)
			XY[index] += XY[index - stride];//index is alway bigger than stride
		__syncthreads();
	}
	// threadIdx.x+1 = 1,2,3,4....
	// stridek index = 1,3,5,7...


	for (unsigned int stride = BLOCK_SIZE / 2; stride > 0; stride /= 2) {
		__syncthreads();
		int index = (threadIdx.x + 1)*stride * 2 - 1;
		if (index < 2 * BLOCK_SIZE)
			XY[index + stride] += XY[index];

	}
	__syncthreads();
	if (i < InputSize) Y[i] = XY[threadIdx.x];
}