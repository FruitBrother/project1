#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include "hip/hip_runtime.h"
#include ""
#include "head.h"
__global__ void join(int *rqa, int *rqb, int* res, dint* equal, int stepa, int stepb, int resstep, int equalsize, int numa, int numb)
{
	int ea, eb;
	bool isMatch = true;
	int k, index, equalid, id;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= numa || y >= numb) return;
	for (k = 0; k < equalsize; k++) {
		ea = equal[k].a + x*stepa;
		eb = equal[k].b + y*stepb;
		if (rqa[ea] != rqb[eb]) {
			isMatch = false;
			break;
		}
	}
	if (isMatch) {
		equalid = 0;
		index = 0;
		//printf("(%d,%d)\n", x, y);
		id = (x*numb + y)*resstep;
		for (k = 0; k < stepa + stepb; k++) {
			if (k < stepa) {
				res[id + k] = rqa[x*stepa + k];
				
				index = k;
			}
			else if (equal[equalid].b != k - stepa) {
				res[id + (++index)] = rqb[y*stepb + (k - stepa)];
			}
			else {
				equalid++;
			}
		}
	}
}

extern "C" void gpu(int *rqa, int *rqb, int* res, dint* equal, int stepa, int stepb, int resstep, int numa, int numb, int equalsize) {
	//allocate memory in gpu and tans the data to it
	int *dev_rqa = 0;
	int *dev_rqb = 0;
	int *dev_res = 0;
	dint *dev_equal = 0;
	hipSetDevice(0);

	hipMalloc((void**)&dev_rqa, numa*stepa * sizeof(int));
	hipMalloc((void**)&dev_rqb, numb*stepb * sizeof(int));
	hipMalloc((void**)&dev_res, numa*numb*resstep * sizeof(int));
	hipMalloc((void**)&dev_equal, sizeof(dint)*MAX(numa, numb));

	hipMemcpy(dev_rqa, rqa, numa*stepa * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_rqb, rqb, numb*stepb * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_equal, equal, sizeof(dint)*MAX(numa, numb), hipMemcpyHostToDevice);

	dim3 block(BLOCKX, BLOCKY);
	dim3 grid((numa + BLOCKX - 1) / BLOCKX, (numb + BLOCKY - 1) / BLOCKY);
	join <<<grid,block>>> (dev_rqa, dev_rqb, dev_res, dev_equal, stepa, stepb, resstep, equalsize, numa, numb);
	hipDeviceSynchronize();
	
	hipMemcpy(res, dev_res, numa*numb*resstep * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_rqa);
	hipFree(dev_rqb);
	hipFree(dev_res);
	hipFree(dev_equal);
}